#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 16

// mandel function in serial.cpp
__device__ int mandel(float c_re, float c_im, int maxIteration) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIteration; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}



__global__ void mandelKernel (float lowerX, float lowerY, float stepX, float stepY, int* d_img, int resX, int resY, int maxIterations){
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    if (thisX < resX && thisY < resY) {
        float x = lowerX + thisX * stepX;
        float y = lowerY + thisY * stepY;
        int index = thisY * resX + thisX;
        d_img[index] = mandel(x, y, maxIterations);
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Allocate device memory
    int* d_img;
    size_t imgSize = resX * resY * sizeof(int);
    hipMalloc((void**)&d_img, imgSize);

    // Define thread block size and grid size
    dim3 threadsPerBlock(N, N);
    dim3 numBlocks((resX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (resY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the kernel
    mandelKernel<<<numBlocks, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, d_img, resX, resY, maxIterations);

    // Copy the result back to host memory
    hipMemcpy(img, d_img, imgSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_img);
}